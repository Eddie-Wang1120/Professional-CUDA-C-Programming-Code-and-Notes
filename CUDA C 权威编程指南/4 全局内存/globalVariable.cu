
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float devData;

__global__ void checkGlobalVariable(){
    //display the original value
    printf("Device: the value of the global variable is %f\n", devData);
    //alter the value
    devData += 2.0f;
}

int main(){
    //initialize the global variable
    float value = 3.14f;
    //value to devData
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
    printf("Host:   copied %f to the global variable\n", value);

    //invoke the kernel
    checkGlobalVariable<<<1, 1>>>();

    //copy the global variable back to the host
    hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float));
    printf("Host:   the value changed by the kernel to %f\n", value);

    hipDeviceReset();
    return EXIT_SUCCESS;
}