
#include <hip/hip_runtime.h>
//nvcc --ptx -o foo.ptx foo.cu

__global__ void intrinsic(float *ptr){
    *ptr = __powf(*ptr, 2.0f);
}

__global__ void standard(float *ptr){
    *ptr = powf(*ptr, 2.0f);
}