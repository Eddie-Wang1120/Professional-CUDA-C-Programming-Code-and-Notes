
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NSTREAM 8

__global__ void sumArrays(float* A, float* B, float* C, int N){
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    int n_repeats = 1000;

    if(idx<N){
        for(int i=0;i<n_repeats;i++){
            C[idx] = A[idx]+B[idx];
        }
    }
}

int main(){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    hipSetDevice(dev);
    printf("device %s starting\n", deviceProp.name);

    hipStream_t *streams = (hipStream_t *)malloc(NSTREAM*sizeof(hipStream_t));
    for(int i=0;i<NSTREAM;i++){
        hipStreamCreate(&(streams[i]));
    }

    int nElem = 1024;
    int nBytes = nElem*sizeof(float);

    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *gpuRef = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);

    hipHostAlloc((float **)&gpuRef, nBytes, hipHostMallocDefault);
    hipHostAlloc((float **)&hostRef, nBytes, hipHostMallocDefault);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    int iElem = nElem / NSTREAM;
    int iBytes = iElem*sizeof(float);

    dim3 grid(1);
    dim3 block(1);

    for(int i = 0;i<NSTREAM;i++){
        int ioffset = i*iElem;
        hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, hipMemcpyHostToDevice, streams[i]);
        sumArrays<<<grid, block, 0, streams[i]>>>(&d_A[ioffset], &d_B[ioffset], &d_C[ioffset], iElem);
    }


    for(int i = 0;i<NSTREAM;i++){
        int ioffset = i*iElem;
        hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes, hipMemcpyDeviceToHost, streams[i]);
    }

    free(h_A);
    free(h_B);    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    hipDeviceReset();

    return EXIT_SUCCESS;

}