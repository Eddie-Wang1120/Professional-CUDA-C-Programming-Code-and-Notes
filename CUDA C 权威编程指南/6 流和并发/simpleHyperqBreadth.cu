
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 30000

int n_streams = 4;

__global__ void kernel_1(){
    double sum = 0.0;
    for(int i=0;i<N;i++){
        sum = sum+tan(0.1)*tan(0.1);
    }
}

__global__ void kernel_2(){
    double sum = 0.0;
    for(int i=0;i<N;i++){
        sum = sum+tan(0.1)*tan(0.1);
    }
}

__global__ void kernel_3(){
    double sum = 0.0;
    for(int i=0;i<N;i++){
        sum = sum+tan(0.1)*tan(0.1);
    }
}

__global__ void kernel_4(){
    double sum = 0.0;
    for(int i=0;i<N;i++){
        sum = sum+tan(0.1)*tan(0.1);
    }
}

int main(int argc, char** argv){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    hipSetDevice(dev);

    printf("device %s starting ...\n", deviceProp.name);

    hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
    for(int i=0;i<n_streams;i++){
        hipStreamCreate(&streams[i]);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(1);
    dim3 grid(1);
    float elapsed_time;
    hipEventRecord(start);
    for(int i = 0; i< n_streams; i++)
    {
    kernel_1<<<grid,block,0,streams[i]>>>();
    }
    for(int i = 0; i< n_streams; i++)
    {
    kernel_2<<<grid,block,0,streams[i]>>>();
    }
    for(int i = 0; i< n_streams; i++)
    {
    kernel_3<<<grid,block,0,streams[i]>>>();
    }
    for(int i = 0; i< n_streams; i++)
    {
    kernel_4<<<grid,block,0,streams[i]>>>();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.3f ms\n",elapsed_time);

    for(int i=0;i<n_streams;i++){
        hipStreamDestroy(streams[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();
    return EXIT_SUCCESS;

}