#include "hip/hip_runtime.h"
#include <stdio.h>
#define DIM 128

void checkResult(int *hostRef, int *gpuRef, const int N){
    double eplison = 1.0E-5;
    int match = 1;
    for(int i=0;i<N;i++){
        if(abs(hostRef[i]-gpuRef[i])>eplison){
            match = 0;
            printf("do not match\n");
            break;
        }
    }

    if(match) printf("match!\n");
    return;

}

void initialData(int* data, int size){
    time_t t;
    srand((unsigned int) time(&t));
    for(int i=0;i<size;i++){
        data[i] = (float)( rand() & 0xFF)/10.0f;
    }
}

__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n){
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockDim.x*blockIdx.x;

    unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx >= n) return;

    if(blockDim.x>=1024 && tid<512) idata[tid] += idata[tid+512];
    __syncthreads();
    if(blockDim.x>=512  && tid<256) idata[tid] += idata[tid+256];
    __syncthreads();
    if(blockDim.x>=256  && tid<128) idata[tid] += idata[tid+128];
    __syncthreads();
    if(blockDim.x>=64   && tid<64 ) idata[tid] += idata[tid+64 ];
    __syncthreads();

    if(tid<32){
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid+32];
        vsmem[tid] += vsmem[tid+16];
        vsmem[tid] += vsmem[tid+8 ];
        vsmem[tid] += vsmem[tid+4 ];
        vsmem[tid] += vsmem[tid+2 ];
        vsmem[tid] += vsmem[tid+1 ];
    }
    if(tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n){
    __shared__ int smem[DIM];
    
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockDim.x*blockIdx.x;

    unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx >= n) return;

    smem[tid] = idata[tid];

    if(blockDim.x>=1024 && tid<512) smem[tid] += smem[tid+512];
    __syncthreads();
    if(blockDim.x>=512  && tid<256) smem[tid] += smem[tid+256];
    __syncthreads();
    if(blockDim.x>=256  && tid<128) smem[tid] += smem[tid+128];
    __syncthreads();
    if(blockDim.x>=64   && tid<64 ) smem[tid] += smem[tid+64 ];
    __syncthreads();

    if(tid<32){
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid+32];
        vsmem[tid] += vsmem[tid+16];
        vsmem[tid] += vsmem[tid+8 ];
        vsmem[tid] += vsmem[tid+4 ];
        vsmem[tid] += vsmem[tid+2 ];
        vsmem[tid] += vsmem[tid+1 ];
    }
    if(tid == 0) g_odata[blockIdx.x] = smem[0];

}

__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n){
    __shared__ int smem[DIM];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x*blockDim.x*4+threadIdx.x;
    int tmpSum = 0;
    if(idx+3*blockDim.x<=n){
        int a1 = g_idata[idx];
        int a2 = g_idata[idx+blockDim.x];
        int a3 = g_idata[idx+2*blockDim.x];
        int a4 = g_idata[idx+3*blockDim.x];
        tmpSum = a1+a2+a3+a4;
    }
    smem[tid] = tmpSum;
    __syncthreads();

    if(blockDim.x>=1024&&tid<512) smem[tid] += smem[tid+512];
    __syncthreads();
    if(blockDim.x>=512&&tid<256)  smem[tid] += smem[tid+256];
    __syncthreads();
    if(blockDim.x>=256&&tid<128)  smem[tid] += smem[tid+128];
    __syncthreads();
    if(blockDim.x>=128&&tid<64 )  smem[tid] += smem[tid+64 ];
    __syncthreads();

    if(tid<32){
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid+32];
        vsmem[tid] += vsmem[tid+16];
        vsmem[tid] += vsmem[tid+8];
        vsmem[tid] += vsmem[tid+4];
        vsmem[tid] += vsmem[tid+2];
        vsmem[tid] += vsmem[tid+1];
    }
    if(tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n){
    extern __shared__ int smem[];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x*blockDim.x*4+threadIdx.x;
    int tmpSum = 0;
    if(idx+3*blockDim.x<=n){
        int a1 = g_idata[idx];
        int a2 = g_idata[idx+blockDim.x];
        int a3 = g_idata[idx+2*blockDim.x];
        int a4 = g_idata[idx+3*blockDim.x];
        tmpSum = a1+a2+a3+a4;
    }
    smem[tid] = tmpSum;
    __syncthreads();

    if(blockDim.x>=1024&&tid<512) smem[tid] += smem[tid+512];
    __syncthreads();
    if(blockDim.x>=512&&tid<256)  smem[tid] += smem[tid+256];
    __syncthreads();
    if(blockDim.x>=256&&tid<128)  smem[tid] += smem[tid+128];
    __syncthreads();
    if(blockDim.x>=128&&tid<64 )  smem[tid] += smem[tid+64 ];
    __syncthreads();

    if(tid<32){
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid+32];
        vsmem[tid] += vsmem[tid+16];
        vsmem[tid] += vsmem[tid+8];
        vsmem[tid] += vsmem[tid+4];
        vsmem[tid] += vsmem[tid+2];
        vsmem[tid] += vsmem[tid+1];
    }
    if(tid == 0) g_odata[blockIdx.x] = smem[0];
}

// __global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n){
//     __shared__ int smem[DIM];
// }

int main(int argc, char** argv){
    int dev = 0;
    hipDeviceProp_t deviceprop;
    hipGetDeviceProperties(&deviceprop, dev);
    hipSetDevice(dev);
    printf("device %s strating...\n", deviceprop.name);

    int nElem = 1<<24;
    size_t nBytes = nElem*sizeof(int);

    int *h_A = (int *)malloc(nBytes);
    int *GmemRef = (int *)malloc(nBytes);
    int *SmemRef = (int *)malloc(nBytes);
    int *SUmemRef = (int *)malloc(nBytes);
    int *SUDmemRef = (int *)malloc(nBytes);
    
    int *d_A, *d_C, *d_D, *d_E, *d_F;
    hipMalloc((int **)&d_A, nBytes);
    hipMalloc((int **)&d_C, nBytes);
    hipMalloc((int **)&d_D, nBytes);
    hipMalloc((int **)&d_E, nBytes);
    hipMalloc((int **)&d_F, nBytes);
    initialData(h_A, nElem);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    
    dim3 block(DIM, 1);
    dim3 grid ((nElem + block.x -1)/block.x, 1);

    reduceGmem<<<grid, block>>>(d_A, d_C, nElem);
    hipMemcpy(d_C, GmemRef, nBytes, hipMemcpyDeviceToHost);

    reduceSmem<<<grid, block>>>(d_A, d_D, nElem);
    hipMemcpy(d_D, SmemRef, nBytes, hipMemcpyDeviceToHost);

    checkResult(GmemRef, SmemRef, nElem);

    reduceSmemUnroll<<<grid.x/4, block>>>(d_A, d_E, nElem);
    hipMemcpy(d_E, SUmemRef, nBytes, hipMemcpyDeviceToHost);

    checkResult(GmemRef, SUmemRef, nElem);

    reduceSmemUnrollDyn<<<grid.x/4, block, DIM*sizeof(int)>>>(d_A, d_F, nElem);
    hipMemcpy(d_F, SUDmemRef, nBytes, hipMemcpyDeviceToHost);

    checkResult(GmemRef, SUDmemRef, nElem);
    hipDeviceReset();
    return EXIT_SUCCESS;

}