#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#define BDIMX 32
#define BDIMY 16


__global__ void setRowReadRow(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y*blockDim.x + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = idx;
    __syncthreads();
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setColReadCol(int *out){
    __shared__ int tile[BDIMX][BDIMY];
    unsigned int idx = threadIdx.y*blockDim.x+threadIdx.x;
    tile[threadIdx.x][threadIdx.y] = idx;
    __syncthreads();
    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadCol(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y*blockDim.x+threadIdx.x;
    
    unsigned int irow = idx/blockDim.y;
    unsigned int icol = idx%blockDim.y;
    
    tile[threadIdx.y][threadIdx.x] = idx;
    __syncthreads();
    out[idx] = tile[icol][irow];
}

__global__ void setRowReadColDyn(int *out){
    extern __shared__ int tile[];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = idx/blockDim.y;
    unsigned int icol = idx%blockDim.y;
    
    unsigned int col_idx = icol*blockDim.x+irow;

    tile[idx] = idx;
    __syncthreads();

    out[idx] = tile[col_idx];
}

__global__ void setRowReadColPad(int *out){
    __shared__ int tile[BDIMY]{BDIMX+1};

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = idx/blockDim.y;
    unsigned int icol = idx%blockDim.y;

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();

    out[idx] = tile[icol][irow];
}

double seconds(){
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char** argv){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    hipSetDevice(dev);
    int iKernel = 0;

    if(argc>1) iKernel = atoi(argv[1]);

    int nElem = BDIMX*BDIMY;
    size_t nBytes = nElem*sizeof(int);

    int *gpuRef = (int *)malloc(nBytes);

    int *d_A;
    hipMalloc((int **)&d_A, nBytes);

    dim3 block(BDIMX, BDIMY);
    dim3 grid(1, 1);

    void (*kernel)(int *);
    char *kernelName;

    switch (iKernel)
    {
    case 0:
        kernel = &setRowReadRow;
        kernelName = "setRowReadRow";
        break;
    
    case 1:
        kernel = &setColReadCol;
        kernelName = "setColReadCol";

    case 2:
        kernel = &setRowReadCol;
        kernelName = "setRowReadCol";

    case 3:
        kernel = &setRowReadColDyn;
        kernelName = "setRowReadColDyn";   

    case 4:
        kernel = &setRowReadColPad;
        kernelName = "setRowReadColPad";       
    }

    double iStart = seconds();
    kernel<<<grid, block>>>(d_A);
    double iElaps = seconds() - iStart;
    printf("%s elapsed %f sec\n",kernelName ,iElaps);

    hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost);

    for(int i=0;i<34;i++){
        printf("%d ", gpuRef[i]);
    }
    
    hipFree(d_A);
    hipFree(gpuRef);

    hipDeviceReset();
    return EXIT_SUCCESS;

}