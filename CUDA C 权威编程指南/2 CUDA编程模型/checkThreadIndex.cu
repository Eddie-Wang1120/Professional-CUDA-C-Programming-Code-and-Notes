
#include <hip/hip_runtime.h>
#include <cstdio>

void initialInt(int *ip, int size){
    for(int i=0;i<size;i++){
        ip[i] = i;
    }
}

void printMatrix(int *C, const int nx, const int ny){
    int *ic = C;
    printf("\nMatrix:%d, %d", nx, ny);
    for(int iy=0;iy<ny;iy++){
        for(int ix=0;ix<nx;ix++){
            printf("%3d\n",ic[ix]);
        }
        ic+=nx;
        printf("\n");
    }
    printf("\n");
}

__global__ void printGPUIdx(int* A, const int nx, const int ny){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    unsigned int idx = iy*nx + ix;

    printf("thread_id (%d,%d) block_id (%d %d) coordinate (%d %d) global index (%d) ival (%d)\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,ix, iy, idx, A[idx]);
}

void sumMatrixOnCPU(float *A, float *B, float*C, const int nx, const int ny){
    float *ia = A;
    float *ib = B;
    float *ic = C;
    for(int iy=0;iy<ny;iy++){
        for(int ix=0;ix<nx;ix++){
            ic[ix] = ia[ix] + ib[ix];
        }
        ic+=nx; ib+=nx; ia+=nx;
    }
}

__global__ void sumMatrixOnGPU(float *Mat_A, float *Mat_B, float *Mat_C, const int nx, const int ny){
    unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
    unsigned idx = iy*nx + ix;

    if(ix < nx && iy < ny)
        Mat_C[idx] = Mat_A[idx] + Mat_B[idx];

}

int main(){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);

    int nx = 8;
    int ny = 6;
    int nxy = nx*ny;
    int nBytes = nxy*(sizeof(float));

    //malloc host mem
    int *h_A;
    h_A = (int *)malloc(nBytes);

    initialInt(h_A, nxy);
    printMatrix(h_A, nx, ny);

    //malloc device mem
    int *d_MatA;
    hipMalloc((void **)&d_MatA, nBytes);

    //transfer
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);

    //set up excution configuration
    dim3 block(4, 2);
    dim3 grid((nx + block.x-1)/block.x, (ny + block.y-1)/block.y);

    //invoke kernel
    printGPUIdx<<< grid, block >>>(d_MatA, nx, ny);
    hipDeviceSynchronize();

    //free host and device
    free(h_A);
    hipFree(d_MatA);

    //reset device
    hipDeviceReset();

    return 0;
}
